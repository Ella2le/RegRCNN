#include "hip/hip_runtime.h"
// ------------------------------------------------------------------
// Faster R-CNN
// Copyright (c) 2015 Microsoft
// Licensed under The MIT License [see fast-rcnn/LICENSE for details]
// Written by Shaoqing Ren
// ------------------------------------------------------------------

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <math.h>
#include <stdio.h>
#include <iostream>
#include <float.h>

#include "sandbox.h"
//tutorial cuda function add

__global__ void add_kernel(float *x, float *y, int n){
    printf("block %d: threadIdx.x %d, threadIdx.y %d, threadIdx.z %d.\n", blockIdx.x, threadIdx.x, threadIdx.y, threadIdx.z);
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i=index; i<n; i+=stride)
        y[i] = x[i] + y[i];
}

void add_cuda(int n=1<<3){
    float *x, *y;
    std::cout << "n: " << n << std::endl;
    hipMallocManaged(&x, n*sizeof(float));
    hipMallocManaged(&y, n*sizeof(float));

    for (int i=0; i<n;i++){
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    int blockSize = 256;
    int numBlocks = getNBlocks(n, blockSize);
    std::cout << "numBlocks " << numBlocks << std::endl;
    add_kernel<<<numBlocks, blockSize>>>(x, y, n);

    hipDeviceSynchronize();

    float maxError = 0.0f;
    for (int i = 0; i < n; i++)
        maxError = fmax(maxError, fabs(y[i]-3.0f));
    std::cout << "Max error: " << maxError << std::endl;

    hipFree(x);
    hipFree(y);

}

/*
__device__ inline float devIoU(float const * const a, float const * const b) {
  float left = fmaxf(a[0], b[0]), right = fminf(a[2], b[2]);
  float top = fmaxf(a[1], b[1]), bottom = fminf(a[3], b[3]);
  float width = fmaxf(right - left + 1, 0.f), height = fmaxf(bottom - top + 1, 0.f);
  float interS = width * height;
  float Sa = (a[2] - a[0] + 1) * (a[3] - a[1] + 1);
  float Sb = (b[2] - b[0] + 1) * (b[3] - b[1] + 1);
  return interS / (Sa + Sb - interS);
}

__global__ void nms_kernel(const int n_boxes, const float nms_overlap_thresh,
                           const float *dev_boxes, unsigned long long *dev_mask) {
  const int row_start = blockIdx.y;
  const int col_start = blockIdx.x;

  // if (row_start > col_start) return;

  const int row_size =
        fminf(n_boxes - row_start * threadsPerBlock, threadsPerBlock);
  const int col_size =
        fminf(n_boxes - col_start * threadsPerBlock, threadsPerBlock);

  __shared__ float block_boxes[threadsPerBlock * 5];
  if (threadIdx.x < col_size) {
    block_boxes[threadIdx.x * 5 + 0] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 0];
    block_boxes[threadIdx.x * 5 + 1] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 1];
    block_boxes[threadIdx.x * 5 + 2] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 2];
    block_boxes[threadIdx.x * 5 + 3] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 3];
    block_boxes[threadIdx.x * 5 + 4] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 4];
  }
  __syncthreads();

  if (threadIdx.x < row_size) {
    const int cur_box_idx = threadsPerBlock * row_start + threadIdx.x;
    const float *cur_box = dev_boxes + cur_box_idx * 5;
    int i = 0;
    unsigned long long t = 0;
    int start = 0;
    if (row_start == col_start) {
      start = threadIdx.x + 1;
    }
    for (i = start; i < col_size; i++) {
      if (devIoU(cur_box, block_boxes + i * 5) > nms_overlap_thresh) {
        t |= 1ULL << i;
      }
    }
    const int col_blocks = DIVUP(n_boxes, threadsPerBlock);
    dev_mask[cur_box_idx * col_blocks + col_start] = t;
  }
}
//
*/

__global__ void nms_kernel(){

}

void nms_cuda(){



}


int main(void){

    nms_cuda();

    return 0;
}